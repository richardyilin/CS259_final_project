#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <sys/time.h>
#include <vector>
#include <set>








using namespace std;








//Define the parameters if not defined externally
#ifndef cmd_def
#define InputNum 2  // Number of input data points (instances)
#define FeatureNum 2  // Number of features in an instance
#define MaxDepth 2  // Number of features in an instance
# define MaxNodeNum (static_cast<int>(pow(2, MaxDepth)) - 1)
#endif
#define VTYPE float
# define DataSize (FeatureNum * InputNum)
# define GainLambda 1
// # define numRows 8
// # define numCols 4
# define MinimumSplitNumInstances 1
# define NUM_THREAD 512



class node
{
public:

    VTYPE predicted_value;
    int node_id;
    int num_instances; // number of instances
    int level;
    int left_child_id;
    int right_child_id;
    double training_loss;
    bool is_leaf;
    int start_index; // start index in data
    int feature_id;
    int feature_index;  // the feature we use to split the node
    int split_index;  // the feature we use to split the node
    VTYPE feature_threshold; // the threshold of the feature value; if it is larger than threshold, it goes to the right child, otherwise the left child

    __host__ __device__ node()
    {
        predicted_value = -1;
        node_id = -1;
        num_instances = -1; // number of instances
        level = -1;
        split_index = -1;
        left_child_id = -1;
        right_child_id = -1;
        training_loss = 0.0;
        is_leaf = false;
        start_index = 0;
        feature_id = -1;
        feature_threshold = 0.0;
        feature_index = -1;
    }
};



#include <utility>


class attribute_id_pair {
public:
   VTYPE attribute_value;
   int instance_id;
};




// read the input file (e.g. ./benchmarks/CASP) return global memory array
// sort the input with attribute values and return
// create y values
// void read_input(string input_path, vector<attribute_id_pair>& data) {




// }
void read_input(attribute_id_pair* data, VTYPE* label) {
// void read_input(string input_path, std::vector<attribute_id_pair>& data, std::vector<VTYPE>& label) {
/*
// include libxl.h for this run//
    Book* book = xlCreateBook();
//  Book*  book = xlCreateBook();
    if (!book) {
        std::cout << "Error creating book." << std::endl;
        return;
    }


    if (book->load(input_path.c_str())) {
        libxl::Sheet* sheet = book->getSheet(0); // Assuming the first sheet
        if (sheet) {
            const int numRows = sheet->lastRow() ;
            const int numCols = sheet->lastCol() + 1;

            data.reserve(numRows*numCols);
            label.reserve(numRows);

        for (int i = 0; i < numCols*numRows; ++i) {
            for (int col = 0; col < numCols; ++col) {
                for (int row = 1; row < numRows; ++row) {
                    attrIdPair.instance_id = i;
                    attribute_id_pair attrIdPair;

                    attrIdPair.attribute_value = (sheet->readNum(row, col));
                    data.push_back(attrIdPair);
                    label.push_back(sheet->readNum(row, 1));
                    }
                }

            }
        }
    }

    else {
        std::cout << "Error loading the workbook." << std::endl;
    }

    book->release();
*/
    // data.reserve(numRows*numCols);
    // label.reserve(numRows);

    // for (int i = 0; i < numCols*numRows; ++i) {
    //     for (int col = 0; col < numCols; ++col) {
    //         for (int row = 1; row < numRows; ++row) {
    //             attribute_id_pair attrIdPair;
    //             attrIdPair.instance_id = i;
    //             attrIdPair.attribute_value = 1;
    //             int label_num;
    //             label_num = i;
        
    //             data.push_back(attrIdPair);
    //             label.push_back(label_num);
    //             }
    //         }  
    //     }
    for (int i = 0; i < FeatureNum; i++) {
        for (int j = 0; j < InputNum; j++) {
            attribute_id_pair pair;
            int id = i * InputNum + j;
            pair.attribute_value = VTYPE(id);
            pair.instance_id = id;
            data[id] = pair;
        }
    }

    for (int i = 0; i < DataSize; i++){
        label[i] = i;
        }
    }














void fill_data(attribute_id_pair* data) {
   for (int i = 0; i < DataSize; i++) {
       attribute_id_pair pair;
       pair.attribute_value = static_cast<VTYPE>(rand() % 101) / 100.0; // Generate a random float between 0 and 1
       pair.instance_id = rand() % 101; // Generate a random integer between 0 and 100
       data[i] = pair;
   }
}
















void fill_label(VTYPE* label){
 int i;
 for (i = 0; i < DataSize; i++){
   label[i] = i;
 }
}








static uint64_t usec;








static __inline__ uint64_t gettime(void) {
 struct timeval tv;
 gettimeofday(&tv, NULL);
 return (((uint64_t)tv.tv_sec) * 1000000 + ((uint64_t)tv.tv_usec));
}








__attribute__ ((noinline))  void begin_roi() {
 usec=gettime();
}
__attribute__ ((noinline))  void end_roi()   {
 usec=(gettime()-usec);
 std::cout << "elapsed (sec): " << usec/1000000.0 << "\n";
}








// __global__ void tree_GPU(node* d_nodes, attribute_id_pair* d_data, VTYPE* d_label, int* d_num_node_eachlevel,int* d_total_num_nodes )
// {   
//     int num_node_this_level = *d_num_node_eachlevel;
//     int total_num_nodes_this_level = *d_total_num_nodes;
//     int total_num_nodes_next_level = total_num_nodes_this_level;
//     int num_node_next_level = 0;
//     printf("d_nodes[0].num_instances %d\n", d_nodes[0].num_instances);
//     printf("num_node_this_level %d, total_num_nodes_this_level %d\n", num_node_this_level, total_num_nodes_this_level);
//     for (int cur_node_id = total_num_nodes_this_level -  num_node_this_level; cur_node_id < total_num_nodes_this_level; cur_node_id++) {
//             printf("cur_node_id %d, total_num_nodes_this_level %d, (cur_node_id < total_num_nodes_this_level) %d\n", cur_node_id, total_num_nodes_this_level, (cur_node_id < total_num_nodes_this_level));
//             node cur_node = d_nodes[cur_node_id];
//             if (cur_node.num_instances < MinimumSplitNumInstances) {
//                 // printf("cur_node.num_instances %d\n", cur_node.num_instances);
//                 // printf("continue\n");
//                 continue;
//             }
//             int num_instances= cur_node.num_instances;
//             int start_index = cur_node.start_index;
//             int node_size = num_instances* FeatureNum;
//             VTYPE best_gain = 0;
//             VTYPE best_split_point = 0;
//             int best_split_index = -1;
//             int best_split_feature_index = 0;
//             VTYPE gamma = 0.0;
//             VTYPE sum_y = 0;
            
//             for (int i = start_index ; i < start_index  + num_instances; i++) {
//                 attribute_id_pair pair = d_data[i];
//                 int instance_id = pair.instance_id;
                
//                 // Access the corresponding label in d_label using the instance_id
//                 VTYPE y = d_label[instance_id];
//                 sum_y += y;
//                 // Perform summation of y or use the value for computation
//                 // ...
//             }
            
//             // Calculate the mean of sum_y as prediction
//             VTYPE prediction = sum_y / num_instances;

//             VTYPE Diff[DataSize];
//             for (int i = start_index ; i < start_index  + node_size; i++) {
//                 attribute_id_pair pair = d_data[i];
//                 int instance_id = pair.instance_id;
                
//                 // Access the corresponding label in d_label using the instance_id
//                 VTYPE y = d_label[instance_id];
//                 Diff[i] = y - prediction;
//                 // Perform summation of y or use the value for computation
//                 // ...
//             }
            
//             // Calculate the prefix sum of Diff
//             VTYPE presum[ DataSize];
//             presum[start_index ] = Diff[start_index ];
//             for (int i = start_index + 1; i < start_index  + node_size; i++) {
//                 presum[i] = presum[i-1] + Diff[i];
//             }

//             for (int j = 0; j < node_size / FeatureNum; j++) {
//                 for (int k = 0; k < FeatureNum; k++) {
//                     int instanceId = k * (node_size / FeatureNum) + j;

//                     // Not the last instance in one attribute
//                     if (((instanceId + 1) % FeatureNum) != 0) {
//                         VTYPE split_point = (d_data[instanceId].attribute_value + d_data[instanceId + 1].attribute_value) / 2;
//                         VTYPE G_l = presum[instanceId];
//                         VTYPE G_r = presum[(k + 1) * (node_size / FeatureNum) - 1] - presum[instanceId];
//                         VTYPE H_l = (instanceId + 1 ) * 2;
//                         VTYPE H_r = (node_size + start_index  - instanceId - 1) * 2;
//                         VTYPE gain = 0.5 * (G_l * G_l / (H_l + GainLambda) + G_r * G_r / (H_r + GainLambda) - (G_l + G_r) * (G_l + G_r) / (H_l + H_r - GainLambda));

//                         if ((gain > best_gain) && (gain > gamma)) {

//                             best_gain = gain;
//                             best_split_point = split_point;
//                             best_split_index = j;
//                             best_split_feature_index = k;
//                         }
//                     }
//                 }
//             }
//             // printf("1\n");
//             if (best_split_index == -1) {
//                 continue;
//             }

//             VTYPE left_instanceid [InputNum];
//             for (int split_index = 0; split_index < best_split_index + 1; split_index++) {
//                 int left_index = best_split_feature_index * (num_instances) + split_index;

//                 attribute_id_pair pair = d_data[left_index];
//                 int original_id = pair.instance_id;
//                 left_instanceid[split_index] = original_id;
//             }

//             // VTYPE counter [DataSize];
//             extern __shared__ int counter [];
//             for (int i = start_index ; i < start_index  + node_size; i++) {
//                 attribute_id_pair pair = d_data[i];
//                 int id = pair.instance_id;
//                 bool found = false;

//                 // Loop through the left indices array
//                 for (int j = 0; j < node_size; j++) {
//                     if (left_instanceid[j] == id) {
//                         found = true;
//                         break;
//                     }
//                 }

//                 // Check if id is found in left_indices
//                 if (found) {
//                     counter[i] = 1;  // Counter for id found in left_indices
//                 } else {
//                     counter[i] = 0;  // Counter for id not found in left_indices
//                 }
//             }
            
//             int getter[2 * FeatureNum] = {0};  // Initialize getter array with 0s
//             int getter_i = 0;

//             for (int i = 0; i < num_instances* FeatureNum; i += num_instances) {
//                 int count_1 = 0;
//                 int count_0 = 0;

//                 for (int j = 0; j < num_instances; j++) {
//                     if (counter[i + j] == 1) {
//                         count_1++;  // Increment occurrence of 1
//                     } else {
//                         count_0++;  // Increment occurrence of 0
//                     }
//                 }

//                 getter[getter_i] = count_1;
//                 getter[getter_i + 1] = count_0;
//                 getter_i += 2;
//             }

//             int getter_group[2 * FeatureNum] = {0};
//             int leftIndex = 0;
//             int rightIndex = FeatureNum;

//             for (int i = 0; i < 2 * FeatureNum; i++) {
//                 if (i % 2 == 0) {
//                     getter_group[leftIndex] = getter[i];
//                     leftIndex++;
//                 } else {
//                     getter_group[rightIndex] = getter[i];
//                     rightIndex++;
//                 }
//             }
            
//             // printf("2\n");
//             int presum_getter[2 * FeatureNum] = {0};  // Initialize presum_getter array with 0s

//             // Calculate prefix sum of getter
//             presum_getter[0] = 0;  // First element is 0
//             for (int i = 1; i < 2 * FeatureNum; i++) {
//                 presum_getter[i] = presum_getter[i - 1] + getter_group[i - 1];
//             }

//             attribute_id_pair sorted_data[DataSize];


//         // Sort data accoring to prefixsum
//             for (int f = 0; f < FeatureNum; f++) {
//                 for (int j = 0; j < num_instances; j++) {
//                     int instanceId = f * num_instances + j;
//                     int offset_left = 0;
//                     int offset_right = 0;
//                     // Check if d_data[instanceId].instance_id is in left_instanceid
//                     bool found = false;
//                     for (int k = 0; k < node_size; k++) {
//                         if (d_data[instanceId].instance_id == left_instanceid[k]) {
//                             found = true;
//                             break;
//                         }
//                     }

//                     // Determine offset and assign values to sorted_data based on found flag
//                     if (found) {
//                         offset_left = presum_getter[f];
//                         sorted_data[offset_left] = d_data[instanceId];
//                         offset_left++;
//                     } else {
//                         offset_right = presum_getter[FeatureNum + f];
//                         sorted_data[offset_right] = d_data[instanceId];
//                         offset_right++;
//                     }
//                 }
//             }
//             memcpy(d_data,sorted_data, sizeof(VTYPE) * DataSize);
//             int new_start_index = -1;

//             // Scan data to find the first element where instance_id is not in left_instanceid
//             for (int i = 0; i < DataSize; i++) {
//                 bool found = false;
//                 for (int j = 0; j < node_size; j++) {
//                     if (d_data[i].instance_id == left_instanceid[j]) {
//                         found = true;
//                         break;
//                     }
//                 }
//                 if (!found) {
//                     new_start_index = i;
//                     break;
//                 }
//             }
//             cur_node.start_index = start_index;
//             cur_node.num_instances = num_instances;
//             // cur_node.level = level ;
//             // cur_node.left_child_id = left_child_id;
//             // cur_node.right_child_id = right_child_id;
//             cur_node.training_loss = best_gain;
//             // cur_node.is_leaf = is_leaf;
//             cur_node.feature_id = best_split_feature_index;
//             cur_node.feature_threshold = best_split_point;
//             cur_node.split_index = best_split_index;

//             // Create the left child node
//             node left_child;
           
//             left_child.start_index = start_index;
//             left_child.num_instances = (new_start_index - start_index)/FeatureNum;
//             left_child.node_id = total_num_nodes_next_level;
//             d_nodes[total_num_nodes_next_level] = left_child;

//             cur_node.left_child_id = total_num_nodes_next_level;
            
//             total_num_nodes_next_level++;
        
//             // left_child .level = level + 1;

//             // Create the right child node
//             node right_child;
//             right_child.start_index = new_start_index;
//             right_child.num_instances = (node_size - new_start_index)/ FeatureNum;
//             right_child.node_id = total_num_nodes_next_level;
//             // right_child.level = level + 1;
//             // Assign left and right child nodes to d_nodes array
//             d_nodes[total_num_nodes_next_level] = right_child;
            
//             cur_node.right_child_id = total_num_nodes_next_level;
//             printf("total_num_nodes_next_level %d\n", total_num_nodes_next_level);
//             total_num_nodes_next_level++;

//             d_nodes[cur_node_id] = cur_node;
//         }
//         *d_total_num_nodes = total_num_nodes_next_level; 
//         *d_num_node_eachlevel = (total_num_nodes_next_level - total_num_nodes_this_level);
// }

__global__ void get_gradient(node* d_nodes, attribute_id_pair* d_data, VTYPE* d_label, VTYPE* d_buffer) {
    int node_id = blockDim.x;
    __shared__ node cur_node = d_nodes[node_id];
    int start_index = cur_node.start_index + threadIdx.x;
    int num_instances = cur_node.num_instances;
    int end_index = start_index + num_instances * FeatureNum;
    int instance_id;
    VTYPE y, y_hat, gradient;

    for (int index = start_index; index < end_index; index += blockDim.x) {
        attribute_id_pair pair = d_data[index];
        instance_id = pair.instance_id;
        y = d_label[instance_id];
        y_hat = cur_node.predicted_value;
        gradient = y_hat - y;
        d_buffer[index] = gradient;
    }

}

int main(void) {
    node nodes[MaxNodeNum];
    node root;
    root.node_id = 0;
    root.num_instances = InputNum;
    root.level = 0;
    root.start_index = 0;
    // ... set other member variables as needed
    nodes[0] = root;
    // Define the aligned array
    
    
    attribute_id_pair data[DataSize];
    VTYPE label [DataSize];
    read_input(data, label);
    cout << "starting program\n";
    // fill_data(data);
    // fill_label(label);


    attribute_id_pair *d_data;
    hipMalloc((void **)(&d_data), sizeof(VTYPE) * DataSize);
    hipMemcpy(d_data, data, sizeof(VTYPE) * DataSize, hipMemcpyHostToDevice);


    VTYPE *d_buffer;
    hipMalloc((void **)(&d_buffer), sizeof(VTYPE) * DataSize);

    VTYPE *d_label;
    hipMalloc((void **)(&d_label), sizeof(VTYPE) * DataSize); // Use d_label instead of label
    hipMemcpy(d_label, label, sizeof(VTYPE) * DataSize, hipMemcpyHostToDevice);

    // dim3 block_size(512); //test, may change in different levels in a tree, calculated by a formula
    // dim3 grid_size(16, 16); //test, may change in different levels in a tree, calculated by a formula
    // Allocate memory on the GPU
    node *d_nodes;
    hipMalloc((void**)&d_nodes, sizeof(node) * MaxNodeNum);
    // Copy the nodes array to the GPU
    hipMemcpy(d_nodes, nodes, sizeof(node) * MaxNodeNum, hipMemcpyHostToDevice);
    


 

    int level = 0;
    int *d_num_node_eachlevel;
    int value = 1;
    int *num_node_this_level = &value;
    hipMalloc((int **)(&d_num_node_eachlevel), sizeof(int)); // Use d_label instead of label
    hipMemcpy(d_num_node_eachlevel, num_node_this_level, sizeof(int), hipMemcpyHostToDevice);

    int *d_total_num_nodes;
    int *total_num_nodes = &value;
    hipMalloc((int **)(&d_total_num_nodes), sizeof(int)); // Use d_label instead of label
    hipMemcpy(d_total_num_nodes, total_num_nodes, sizeof(int), hipMemcpyHostToDevice);

    dim3 block_size, thread_size;
    while (level < MaxDepth - 1) {
        printf("level %d\n", level);
        block_size = (*num_node_this_level);
`       thread_size = (NUM_THREAD);
        get_gradient<<<block_size, thread_size>>>(d_nodes, d_data, d_label, d_buffer);
        hipDeviceSynchronize();
        hipMemcpy(data, d_data, sizeof(attribute_id_pair)* DataSize, hipMemcpyDeviceToHost);
        hipMemcpy(nodes, d_nodes, sizeof(node)* MaxNodeNum, hipMemcpyDeviceToHost);
        hipMemcpy(num_node_this_level, d_num_node_eachlevel, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(total_num_nodes, d_total_num_nodes, sizeof(int), hipMemcpyDeviceToHost);
        level++;
    }

    // begin_roi();
    // classifier_GPU<<<grid_size, block_size>>>(d_data);
    // hipDeviceSynchronize();
    // end_roi();

for (int i = 0; i < MaxNodeNum; i++) {
    std::cout << "Node " << i << ":" << std::endl;
    std::cout << "predicted_value: " << nodes[i].predicted_value << std::endl;
    std::cout << "node_id: " << nodes[i].node_id << std::endl;
    std::cout << "num_instances: " << nodes[i].num_instances << std::endl;
    std::cout << "level: " << nodes[i].level << std::endl;
    std::cout << "left_child_id: " << nodes[i].left_child_id << std::endl;
    std::cout << "right_child_id: " << nodes[i].right_child_id << std::endl;
    std::cout << "training_loss: " << nodes[i].training_loss << std::endl;
    std::cout << "is_leaf: " << nodes[i].is_leaf << std::endl;
    std::cout << "start_index: " << nodes[i].start_index << std::endl;
    std::cout << "feature_id: " << nodes[i].feature_id << std::endl;
    std::cout << "feature_threshold: " << nodes[i].feature_threshold << std::endl;
    std::cout << "split_index: " << nodes[i].split_index << std::endl;
    std::cout << "-------------------------" << std::endl;
}
    hipFree(d_nodes);
    hipFree(d_data);
    hipFree(d_label);
}



























